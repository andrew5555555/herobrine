
#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <chrono>
#include <string>
using namespace std;
#include <stdio.h>
#include <assert.h>
#include <math.h>
#define ll long long int


const ll m = 0x5DEECE66Dll;
const ll mask = (1ll << 48) - 1;

#define advance1(s) s = (s * m + 11ll) & mask
#define advance3759(s) s = (s*0x6fe85c031f25ll + 0x8f50ecff899ll)&mask
#define advance16(s) s = (s*0x6dc260740241ll + 0xd0352014d90ll)&mask
#define advance387(s) s = (s*0x5fe2bcef32b5ll + 0xb072b3bf0cbdll)&mask
#define advance774(s) s = (s*0xf8d900133f9ll + 0x5738cac2f85ell)&mask
#define advance11(s) s = (s*0x53bce7b8c655ll + 0x3bb194f24a25ll)&mask
#define advance3(s) s = (s*0xd498bd0ac4b5ll + 0xaa8544e593dll)&mask
#define advance17(s) s = (s*0xee96bd575badll + 0xc45d76fd665bll)&mask

#define regress1(s) s = (s*0xdfe05bcb1365ll + 0x615c0e462aa9ll)&mask
#define regress3(s) s = (s*0x13a1f16f099dll + 0x95756c5d2097ll)&mask
#define regress3759(s) s = (s*0x63a9985be4adll + 0xa9aa8da9bc9bll)&mask
#define advance2(s) s = (s*0xbb20b4600a69ll + 0x40942de6ball)&mask

#define getNextInt(x, s) advance1(s); x = (int)(s>>16)

// need spare longs temp1 and temp2
#define getNextLong(x, s) getNextInt(temp1, s); getNextInt(temp2, s); x = (temp1 << 32) + temp2

#define getIntBounded(x, s, n) if ((n&(-n))==n) {advance1(s); x = (int)((n*(s>>17)) >> 31);} else {do{advance1(s); bits = s>>17; val = bits%n;}while(bits-val+(n-1)<0); x=val;}

#define getBits(x, s, n) advance1(s); x = (int) (s >> (48-n));

#define setSeed(s, x) s = x^m&mask


#define MAX_TREES 12 // can change this later (performance & output is not very sensitive to this parameter)

#define x_1 9
#define z_1 2
const int x_2 = x_1 - 7;
const int z_2 = z_1 + 1;

#define SMALL_TREE_SPACING 2
#define BIG_TREE_SPACING 7 

__device__ __managed__ unsigned long long int num_found = 0;
#define memsz 100000
__device__ __managed__ ll ret[memsz];

__device__ __managed__ char table[16][16]; // can a tree spawn here
// 0 : no
// 1 : tree_1's territory
// 2 : tree_2's territory
// 3 : joint tree_1 and tree_2
// 8 : a small tree's leaves
// 9 : I don't know (assume yes)
__device__ __managed__ int visited = 0;



__device__ void output_seed(ll s) {
	regress1(s);
	ll id = atomicAdd(&num_found, 1ull); // dw about red underline
	ret[id] = s;
}

// coordinates are relative to tree_1 position
void add_unseen(int sz, int ez, int x) {
	sz -= 2; // fudge factor for safety
	int i = x + x_1;
	for (int j = sz + z_1; j < ez + z_1; j++) {
		if ((i & 15) == i && (j & 15) == j)
		{
			table[i][j] = 9;
		}
	}
}

// fill a rectangle with a number (1 or 2 or 3)
void fill_rect(int sx, int sz, int ex, int ez, int fill) {
	assert(fill == 1 || fill == 2 || fill == 3);
	for (int i = max(0, x_1 + sx); i < min(16, x_1 + ex); i++) {
		for (int j = max(0, z_1 + sz); j < min(16, z_1 + ez); ++j)
		{
			table[i][j] = fill;
		}
	}
}

void init_table() {
	for (int i = 0; i < 16; ++i)
	{
		for (int j = 0; j < 16; ++j)
		{
			table[i][j] = 0;
		}
	}
	/*
	// big tree leaves
	fill_rect(-1, -2, 5, 10, 1);
	fill_rect(-6, -2, -1, 10, 3);
	fill_rect(-13, 0, -6, 10, 2);
	// small pieces of tree:
	fill_rect(1, -3, 4, -2, 1);
	fill_rect(5, 2, 6, 3, 1);
	fill_rect(5, 5, 6, 7, 1);
	fill_rect(-8, -1, -6, 0, 2);
	*/
	for (int i = max(0, x_1 - BIG_TREE_SPACING); i <= min(15, x_1 + BIG_TREE_SPACING); ++i)
	{
		for (int j = max(0, z_1 - BIG_TREE_SPACING); j <= min(15, z_1 + BIG_TREE_SPACING); ++j)
		{
			table[i][j] |= 1;
		}
	}
	for (int i = max(0, x_2 - BIG_TREE_SPACING); i <= min(15, x_2 + BIG_TREE_SPACING); ++i)
	{
		for (int j = max(0, z_2 - BIG_TREE_SPACING); j <= min(15, z_2 + BIG_TREE_SPACING); ++j)
		{
			table[i][j] |= 2;
		}
	}

	// add locations where it's unclear if there is a tree or not



	add_unseen(4, 14, 6);
	add_unseen(7, 14, 5);
	add_unseen(8, 14, 4);
	add_unseen(9, 14, 3);
	add_unseen(10, 14, 2);
	add_unseen(11, 14, 1);
	add_unseen(12, 14, 0);
	add_unseen(12, 14, -1);
	add_unseen(12, 14, -2);
	add_unseen(13, 14, -3);
	add_unseen(13, 14, -4);
	add_unseen(13, 14, -5);
	add_unseen(14, 14, -6);
	add_unseen(14, 14, -7);
	add_unseen(14, 14, -8);
	add_unseen(14, 14, -9);
	add_unseen(14, 14, -10);
	add_unseen(14, 14, -11);
	add_unseen(14, 14, -12);
	add_unseen(14, 14, -13);

	// near a small tree 3
	int x_3 = x_1 - 12;
	int z_3 = z_1 - 7;
	for (int i = max(0, x_3 - SMALL_TREE_SPACING); i <= min(15, x_3 + SMALL_TREE_SPACING); ++i)
	{
		for (int j = max(0, z_3 - SMALL_TREE_SPACING); j <= min(15, z_3 + SMALL_TREE_SPACING); ++j)
		{
			table[i][j] = 8;
		}
	}
	// near a small tree 4
	int x_4 = x_1 - 10;
	int z_4 = z_1 - 12;
	for (int i = max(0, x_4 - SMALL_TREE_SPACING); i <= min(15, x_4 + SMALL_TREE_SPACING); ++i)
	{
		for (int j = max(0, z_4 - SMALL_TREE_SPACING); j <= min(15, z_4 + SMALL_TREE_SPACING); ++j)
		{
			table[i][j] = 8;
		}
	}
	// near a small tree 5
	int x_5 = x_1 - 5;
	int z_5 = z_1 - 15;
	for (int i = max(0, x_5 - SMALL_TREE_SPACING); i <= min(15, x_5 + SMALL_TREE_SPACING); ++i)
	{
		for (int j = max(0, z_5 - SMALL_TREE_SPACING); j <= min(15, z_5 + SMALL_TREE_SPACING); ++j)
		{
			table[i][j] = 8;
		}
	}

	for (int i = 15; i >= 0; --i)
	{
		for (int j = 0; j < 16; ++j)
		{
			printf("%d ", table[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}




int file_num = 0;
ofstream get_next_file() {
	string path("_inter21/_intermediate");
	path = path + to_string(file_num++) + ".txt";
	ofstream of(path);
	return of;
}


// todo: work this out
__device__ int is_field_878_e_ok(int field_878_e) {
	//return 1;
	return field_878_e == 11;// || field_878_e == 12;
	//return field_878_e >= 11 && field_878_e <= 12;
}

// we get a 48-bit candidate; (it's given that this is a big tree chunk) 
__device__ void check_tree_seed(ll s, char s_table[16][16]) {
	ll original = s;
	int found_1 = 0;
	int found_2 = 0;
	int tree_x, tree_z;
	int bits, val;
	ll temp1, temp2;
	ll saved_seed;
	for (int i = 0; i < MAX_TREES; ++i) {
		getBits(tree_x, s, 4);
		getBits(tree_z, s, 4);
		if (i == 0) {
			saved_seed = s; // save for tree-height check
		}
		advance2(s);

		if (!found_1 && tree_x == x_1 && tree_z == z_1) {
			if (found_2) {
				output_seed(original);
				return;
			}
			found_1 = 1;
		}
		else if (!found_2 && tree_x == x_2 && tree_z == z_2) {
			if (found_1) {
				output_seed(original);
				return;
			}
			found_2 = 1;
		}
		else {
			char lookup = s_table[tree_x][tree_z];
			if ((lookup == 0) || (lookup == 1 && !found_1) || (lookup == 2 && !found_2) || (lookup == 3 && !found_1 && !found_2)) {
				return; // seed is eliminated
			}
		}
		if (i == 0) {
			// we do this check as late as possible because it is more expensive
			ll internal_big_tree_seed;
			getNextLong(internal_big_tree_seed, saved_seed);
			setSeed(internal_big_tree_seed, internal_big_tree_seed);
			int field_878_e;
			getIntBounded(field_878_e, internal_big_tree_seed, 12);
			if (!is_field_878_e_ok(field_878_e + 5)) {
				return; // wrong tree-size
			}
		}

	}
}

__global__ void treeKernel(ll global_id) {
	__shared__ char s_table[16][16];
	ll tid = threadIdx.x;
	s_table[tid % 16][tid / 16] = table[tid % 16][tid / 16];
	__syncthreads();
	ll bid = blockIdx.x;
	

	ll div_ten = ((global_id << 25) | (bid << 17));
	ll upper31 = mask & (10ll*div_ten); // upper 31 bits of seed are multiple of ten
	if (upper31 < div_ten) {
		return; // overflowed the 48 bits (happens on final few blocks)
	}
	ll upper39 = upper31 | (tid << 9);

	for (ll lower9 = 0; lower9 < (1ll << 9); lower9++) {
		ll seed = upper39 | lower9;
		check_tree_seed(seed, s_table);
	}
 }


#define RUN_ID 200
hipError_t do_work() {

	
	ofstream log("big_tree_log_200.txt");
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}




	int threads_per_block = 256;
	int num_blocks = 256;        // can't change these without breaking code

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "big tree kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	//ll num_found = 0;
	printf("begin xyz\n");
	auto start = chrono::steady_clock::now();
	int num_written = 0;
	ll NUM_ITERS = 838861; // ceil(2^31 / 2^8 / 10)
	//NUM_ITERS = 10000;
	for (ll o = 0; o < NUM_ITERS; o ++) {
		treeKernel <<<num_blocks, threads_per_block >>> (o);
		if (o % 100 == 0) {
			ofstream fout = get_next_file();
			hipDeviceSynchronize();
			fout << RUN_ID << endl;
			fout << x_1 << endl << z_1 << endl;
			fout << num_found << endl;
			for (int i = 0; i < num_found; i++) {
				fout << ret[i] << endl;
				num_written++;
			}
			fout.close();
			num_found = 0;
			//printf("%lld\n", o);
			auto end = chrono::steady_clock::now();
			ll time = (chrono::duration_cast<chrono::microseconds>(end - start).count());
			float eta = ((838861-o) / ((float)o)) * ((float)time) / 3600.0 / 1000000.0;
			log << "doing " << o << " time taken us =" << time << " eta (hrs) = " << eta << endl;
			log.flush();
		}
		

	}
	
	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda not sync: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	auto end = chrono::steady_clock::now();
	cout << "time taken us =" << chrono::duration_cast<chrono::microseconds>(end - start).count() << endl;

	ofstream fout = get_next_file();
	fout << RUN_ID << endl;
	fout << x_1 << endl << z_1 << endl;
	fout << num_found << endl;
	for (int i = 0; i < num_found; i++) {
		fout << ret[i] << endl;
		num_written++;
	}
	fout.close();
	cout << "total seeds written=" << num_written << endl;

	

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching big tree kernel!\n", cudaStatus);
	}

	return cudaStatus;
}


int main()
{
	init_table();
	// Add vectors in parallel.
	hipError_t cudaStatus = do_work();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda failed!");
		return 1;
	}
	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}